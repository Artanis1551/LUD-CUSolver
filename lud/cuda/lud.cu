/*
 * =====================================================================================
 *
 *       Filename:  lud.cu
 *
 *    Description:  The main wrapper for the suite
 *
 *        Version:  1.0
 *        Created:  10/22/2009 08:40:34 PM
 *       Revision:  none
 *       Compiler:  gcc
 *
 *         Author:  Liang Wang (lw2aw), lw2aw@virginia.edu
 *        Company:  CS@UVa
 *
 * =====================================================================================
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <getopt.h>
#include <stdlib.h>
#include <assert.h>

#include "common.h"

//Defines kernel sizes (block size)
#ifdef RD_WG_SIZE_0_0
        #define BLOCK_SIZE RD_WG_SIZE_0_0
#elif defined(RD_WG_SIZE_0)
        #define BLOCK_SIZE RD_WG_SIZE_0
#elif defined(RD_WG_SIZE)
        #define BLOCK_SIZE RD_WG_SIZE
#else
        #define BLOCK_SIZE 16
#endif

//initializes do_verify
static int do_verify = 0;
static int cuSolve = 0;

//Options for matrix
static struct option long_options[] = {
  /* name, has_arg, flag, val */
  {"input", 1, NULL, 'i'},
  {"size", 1, NULL, 's'},
  {"cuSolver", 0, NULL, 'c'},
  {"verify", 0, NULL, 'v'},
  {0,0,0,0}
};

//Creates function for kernel launch
extern void
lud_cuda(double *d_m, int matrix_dim, int cuSolve);


int
main ( int argc, char *argv[] )
{
  printf("WG size of kernel = %d X %d\n", BLOCK_SIZE, BLOCK_SIZE);

  int matrix_dim = 32; /* default matrix_dim */
  int opt, option_index = 1;
  func_ret_t ret;
  const char *input_file = NULL;
  double *d_m, *mm;
  double *m;
  stopwatch sw;

  //Gets option from run call (loops for all options)
  while ((opt = getopt_long(argc, argv, "::vcs:i:", 
                            long_options, &option_index)) != -1 ) {
    switch(opt){
    //if option -i read input values from input file
    case 'i':
      input_file = optarg;
      break;
    //if option -v enable verification
    case 'v':
      do_verify = 1;
      break;
    //Manually sets matrix_dim to value in run call
    case 's':
      matrix_dim = atoi(optarg);
      printf("Generate input matrix internally, size =%d\n", matrix_dim);
      // fprintf(stderr, "Currently not supported, use -i instead\n");
      // fprintf(stderr, "Usage: %s [-v] [-s matrix_size|-i input_file]\n", argv[0]);
      // exit(EXIT_FAILURE);
      break;
    case 'c':
      cuSolve = 1;
    break;
    //Indicates that option is invalid
    case '?':
      fprintf(stderr, "invalid option\n");
      break;
    //Indicates that argument is missing
    case ':':
      fprintf(stderr, "missing argument\n");
      break;
    //Tells user how to use options
    default:
      fprintf(stderr, "Usage: %s [-v] [-s matrix_size|-i input_file]\n",
	      argv[0]);
      exit(EXIT_FAILURE);
    }
  }
  
  //If any elements are left that are not options it prints how to use options
  if ( (optind < argc) || (optind == 1)) {
    fprintf(stderr, "Usage: %s [-v] [-s matrix_size|-i input_file]\n", argv[0]);
    exit(EXIT_FAILURE);
  }

  //If option was -i this code calls a function that reads a file. and prints if read failed
  if (input_file) {
    printf("Reading matrix from file %s\n", input_file);
    ret = create_matrix_from_file(&m, input_file, &matrix_dim);
    if (ret != RET_SUCCESS) {
      m = NULL;
      fprintf(stderr, "error create matrix from file %s\n", input_file);
      exit(EXIT_FAILURE);
    }
  } //If input_file is not set it creates the matrix
  else if (matrix_dim) {
    printf("Creating matrix internally size=%d\n", matrix_dim);
    ret = create_matrix(&m, matrix_dim);
    if (ret != RET_SUCCESS) {
      m = NULL;
      fprintf(stderr, "error create matrix internally size=%d\n", matrix_dim);
      exit(EXIT_FAILURE);
    }
  }

  //Else say no input file was specified
  else {
    printf("No input file specified!\n");
    exit(EXIT_FAILURE);
  }

  //If do_verify is 1 print matrix before LUD and duplicate matrix
  if (do_verify){
    //printf("Before LUD\n");
    //print_matrix(m, matrix_dim);
    matrix_duplicate(m, &mm, matrix_dim);
  }

  //Allocate memory and begin timing and copy memory
  hipMalloc((void**)&d_m, 
             matrix_dim*matrix_dim*sizeof(double));

  /* beginning of timing point */
  stopwatch_start(&sw);
  hipMemcpy(d_m, m, matrix_dim*matrix_dim*sizeof(double), 
	     hipMemcpyHostToDevice);

  //Kernel launch
  lud_cuda(d_m, matrix_dim, cuSolve);
  

  //Copy back memory
  hipMemcpy(m, d_m, matrix_dim*matrix_dim*sizeof(double), 
	     hipMemcpyDeviceToHost);

  /* end of timing point */
  stopwatch_stop(&sw);
  printf("Time consumed(ms): %lf\n", 1000*get_interval_by_sec(&sw));

  hipFree(d_m);

  //If do_verify is 1 print matrix after LUD and verify lud by comparing gpu implementation with cpu implementation
  if (do_verify){
    //printf("After LUD\n");
    //print_matrix(m, matrix_dim);
    printf(">>>Verify<<<<\n");
    lud_verify(mm, m, matrix_dim, cuSolve); 
    free(mm);
  }

  free(m);

  return EXIT_SUCCESS;
}				/* ----------  end of function main  ---------- */
